#pragma once

#include <cubvh/api.h>

#include <cubvh/common.h>
#include <cubvh/bvh.cuh>

#include <Eigen/Dense>

using namespace Eigen;

using Verts = Matrix<float, Dynamic, 3, RowMajor>;
using Trigs = Matrix<uint32_t, Dynamic, 3, RowMajor>;

namespace cubvh {

class cuBVHImpl : public cuBVH {
public:

    // accept numpy array (cpu) to init 
    cuBVHImpl(Ref<const Verts> vertices, Ref<const Trigs> triangles) : cuBVH() {

        const size_t n_vertices = vertices.rows();
        const size_t n_triangles = triangles.rows();

        triangles_cpu.resize(n_triangles);

        for (size_t i = 0; i < n_triangles; i++) {
            triangles_cpu[i] = {vertices.row(triangles(i, 0)), vertices.row(triangles(i, 1)), vertices.row(triangles(i, 2)), (int64_t)i};
        }

        if (!triangle_bvh) {
            triangle_bvh = TriangleBvh::make();
        }

        triangle_bvh->build(triangles_cpu, 8);

        triangles_gpu.resize_and_copy_from_host(triangles_cpu);

        // TODO: need OPTIX
        // triangle_bvh->build_optix(triangles_gpu, m_inference_stream);

    }

    void ray_trace(at::Tensor rays_o, at::Tensor rays_d, at::Tensor positions, at::Tensor face_id, at::Tensor depth) {

        const uint32_t n_elements = rays_o.size(0);
        hipStream_t stream = at::cuda::getCurrentCUDAStream();

        triangle_bvh->ray_trace_gpu(n_elements, rays_o.data_ptr<float>(), rays_d.data_ptr<float>(), positions.data_ptr<float>(), face_id.data_ptr<int64_t>(), depth.data_ptr<float>(), triangles_gpu.data(), stream);
    }

    void unsigned_distance(at::Tensor positions, at::Tensor distances, at::Tensor face_id, at::optional<at::Tensor> uvw) {

        const uint32_t n_elements = positions.size(0);
        hipStream_t stream = at::cuda::getCurrentCUDAStream();

        triangle_bvh->unsigned_distance_gpu(n_elements, positions.data_ptr<float>(), distances.data_ptr<float>(), face_id.data_ptr<int64_t>(), uvw.has_value() ? uvw.value().data_ptr<float>() : nullptr, triangles_gpu.data(), stream);

    }

    void signed_distance(at::Tensor positions, at::Tensor distances, at::Tensor face_id, at::optional<at::Tensor> uvw, uint32_t mode) {

        const uint32_t n_elements = positions.size(0);
        hipStream_t stream = at::cuda::getCurrentCUDAStream();

        triangle_bvh->signed_distance_gpu(n_elements, mode, positions.data_ptr<float>(), distances.data_ptr<float>(), face_id.data_ptr<int64_t>(), uvw.has_value() ? uvw.value().data_ptr<float>() : nullptr, triangles_gpu.data(), stream);
    }

    std::vector<Triangle> triangles_cpu;
    GPUMemory<Triangle> triangles_gpu;
    std::shared_ptr<TriangleBvh> triangle_bvh;
};
    
cuBVH* create_cuBVH(Ref<const Verts> vertices, Ref<const Trigs> triangles) {
    return new cuBVHImpl{vertices, triangles};
}

} // namespace cubvh